#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t, typename bound_t>
__device__ __forceinline__ scalar_t clamp(const scalar_t v, const bound_t lo, const bound_t hi) {
  return min(max(v, lo), hi);
}

template <typename scalar_t, bool dense_mode>
__global__ void total_variation_2d_add_grad_cuda_kernel(
    const scalar_t* __restrict__ param,
    scalar_t* __restrict__ grad,
    float wx, float wy, 
    const size_t sz_i, const size_t sz_j, const size_t N) {

  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index<N && (dense_mode || grad[index]!=0)) {
    // const size_t k = index % sz_k;
    // const size_t j = index / sz_k % sz_j;
    // const size_t i = index / sz_k / sz_j % sz_i;
    const size_t j = index / sz_j;
    const size_t i = index / sz_j % sz_i;

    float grad_to_add = 0;
    // grad_to_add += (k==0      ? 0 : wz * clamp(param[index]-param[index-1], -1.f, 1.f));
    // grad_to_add += (k==sz_k-1 ? 0 : wz * clamp(param[index]-param[index+1], -1.f, 1.f));
    // grad_to_add += (j==0      ? 0 : wy * clamp(param[index]-param[index-sz_k], -1.f, 1.f));
    // grad_to_add += (j==sz_j-1 ? 0 : wy * clamp(param[index]-param[index+sz_k], -1.f, 1.f));
    // grad_to_add += (i==0      ? 0 : wz * clamp(param[index]-param[index-sz_k*sz_j], -1.f, 1.f));
    // grad_to_add += (i==sz_i-1 ? 0 : wz * clamp(param[index]-param[index+sz_k*sz_j], -1.f, 1.f));
    grad_to_add += (j==0      ? 0 : wy * clamp(param[index]-param[index-1], -1.f, 1.f));
    grad_to_add += (j==sz_j-1 ? 0 : wy * clamp(param[index]-param[index+1], -1.f, 1.f));
    grad_to_add += (i==0      ? 0 : wx * clamp(param[index]-param[index-sz_j], -1.f, 1.f));
    grad_to_add += (i==sz_i-1 ? 0 : wx * clamp(param[index]-param[index+sz_j], -1.f, 1.f));


    grad[index] += grad_to_add;
  }
}

void total_variation_2d_add_grad_cuda(torch::Tensor param, torch::Tensor grad, float wx, float wy, bool dense_mode) {
  const size_t N = param.numel();
  const size_t sz_i = param.size(2);
  const size_t sz_j = param.size(3);
  // const size_t sz_k = param.size(4);
  const int threads = 256;
  const int blocks = (N + threads - 1) / threads;

  wx /= 4;
  wy /= 4;
  // wz /= 6;

  if(dense_mode) {
    AT_DISPATCH_FLOATING_TYPES(param.type(), "total_variation_2d_add_grad_cuda", ([&] {
      total_variation_2d_add_grad_cuda_kernel<scalar_t,true><<<blocks, threads>>>(
          param.data<scalar_t>(),
          grad.data<scalar_t>(),
          wx, wy,
          sz_i, sz_j, N);
    }));
  }
  else {
     AT_DISPATCH_FLOATING_TYPES(param.type(), "total_variation_2d_add_grad_cuda", ([&] {
      total_variation_2d_add_grad_cuda_kernel<scalar_t,false><<<blocks, threads>>>(
          param.data<scalar_t>(),
          grad.data<scalar_t>(),
          wx, wy,
          sz_i, sz_j, N);
    }));
  }
}

